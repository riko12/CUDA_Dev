// vector_add.cu
#include <hip/hip_runtime.h>
#include <stdio.h>

// CUDA Kernel for vector addition
__global__ void vectorAdd(const float* a, const float* b, float* c, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        c[idx] = a[idx] + b[idx];
    }
}

// CUDA function wrapper that can be called from C++
extern "C" bool cuda_vector_add(const float* h_a, const float* h_b, float* h_c, int n) {
    float *d_a, *d_b, *d_c;
    
    // Allocate device memory
    hipError_t err = hipMalloc(&d_a, n * sizeof(float));
    if (err != hipSuccess) return false;
    err = hipMalloc(&d_b, n * sizeof(float));
    if (err != hipSuccess) {
        hipFree(d_a);
        return false;
    }
    err = hipMalloc(&d_c, n * sizeof(float));
    if (err != hipSuccess) {
        hipFree(d_a);
        hipFree(d_b);
        return false;
    }

    // Copy inputs to device
    hipMemcpy(d_a, h_a, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, n * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel
    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;
    vectorAdd<<<numBlocks, blockSize>>>(d_a, d_b, d_c, n);

    // Copy result back to host
    hipMemcpy(h_c, d_c, n * sizeof(float), hipMemcpyDeviceToHost);

    // Cleanup
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return true;
}